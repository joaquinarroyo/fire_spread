#include "hip/hip_runtime.h"
#include "spread_functions.cuh"

#define _USE_MATH_DEFINES
#include <cmath>
#include <random>
#include <vector>
#include <omp.h>
#include <iostream>
#include <array>
#include <random>

#include "fires.hpp"
#include "landscape.hpp"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

struct FireKernelParams {
    const float* elevation;
    const float* fwi;
    const float* aspect;
    const float* wind_dir;
    const float* vegetation_type;
    const uint8_t* burnable;

    int* burned_bin;
    const int width;
    const int height;

    unsigned int* processed_cells;
    
    const SimulationParams* params;

    const float distance;
    const float upper_limit;
    const float elevation_mean;
    const float elevation_sd;

    const int rng_seed;
};

constexpr float PIf = 3.1415927f;
float h_angles[8] = {
    PIf * 3 / 4, PIf, PIf * 5 / 4, PIf / 2,
    PIf * 3 / 2, PIf / 4, 0, PIf * 7 / 4
};
int h_moves[8][2] = {
    { -1, -1 }, { -1, 0 }, { -1, 1 }, { 0, -1 },
    { 0, 1 }, { 1, -1 }, { 1, 0 }, { 1, 1 }
};
__constant__ float d_angles[8];
__constant__ int d_moves[8][2];

__global__ void init_rng_kernel(hiprandState* states, int width, int height, int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = width * height;
    if (tid < total) {
        int i = tid % width;
        int j = tid / width;
        unsigned long long cell_seed = seed ^ (i * 73856093) ^ (j * 19349663);
        hiprand_init(cell_seed, 0, 0, &states[tid]);
    }
}


__device__ void spread_probability_device(
    float burning_elevation,
    float burning_wind_direction,
    const float* elevations,
    const float* vegetation_types,
    const float* fwis,
    const float* aspects,
    const float* upper_limits,
    const SimulationParams* params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float* probs_out
) {
    for (int n = 0; n < 8; n++) {
        float slope_term = __sinf(atanf((elevations[n] - burning_elevation) / distance));
        float wind_term = __cosf(d_angles[n] - burning_wind_direction);
        float elev_term = (elevations[n] - elevation_mean) / elevation_sd;

        float linpred = params->independent_pred;

        if ((int)vegetation_types[n] == SUBALPINE) {
            linpred += params->subalpine_pred;
        } else if ((int)vegetation_types[n] == WET) {
            linpred += params->wet_pred;
        } else if ((int)vegetation_types[n] == DRY) {
            linpred += params->dry_pred;
        }

        linpred += params->fwi_pred * fwis[n];
        linpred += params->aspect_pred * aspects[n];
        linpred += wind_term * params->wind_pred + elev_term * params->elevation_pred + slope_term * params->slope_pred;

        probs_out[n] = upper_limits[n] / (1.0f + __expf(-linpred));
    }
}

__global__ void fire_persistent_kernel(
    FireKernelParams args,
    int* frontier_0, int* frontier_1,
    int* frontier_size,
    int* next_frontier_0, int* next_frontier_1,
    int* next_frontier_count,
    int* done_flag,
    int* in_next_frontier,
    hiprandState* rng_states
) {
    const float* elevation = args.elevation;
    const float* fwi = args.fwi;
    const float* aspect = args.aspect;
    const float* wind_dir = args.wind_dir;
    const float* vegetation_type = args.vegetation_type;
    const uint8_t* burnable = args.burnable;

    int* burned_bin = args.burned_bin;
    int width = args.width;
    int height = args.height;
    const SimulationParams* params = args.params;

    float distance = args.distance;
    float upper_limit = args.upper_limit;
    float elevation_mean = args.elevation_mean;
    float elevation_sd = args.elevation_sd;

    unsigned int local_processed_cells = 0;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (!*done_flag) {
        int frontier_len = *frontier_size;

        for (int idx = tid; idx < frontier_len; idx += gridDim.x * blockDim.x) {
            int i = frontier_0[idx];
            int j = frontier_1[idx];
            int center_idx = j * width + i;

            hiprandState local_state = rng_states[center_idx];

            float elev_c = elevation[center_idx];
            float wind_c = wind_dir[center_idx];

            int n_coords_0[8], n_coords_1[8];
            int n_indices[8];
            uint8_t n_out_flags[8];
            float n_elev[8], n_fwi[8], n_asp[8], n_veg[8], n_burn[8], n_upper[8];

            for (int n = 0; n < 8; ++n) {
                int ni = i + d_moves[n][0];
                int nj = j + d_moves[n][1];
                n_coords_0[n] = ni;
                n_coords_1[n] = nj;

                if (ni < 0 || nj < 0 || ni >= width || nj >= height) {
                    n_out_flags[n] = 1;
                    n_indices[n] = 0;
                    n_elev[n] = n_fwi[n] = n_asp[n] = n_veg[n] = 0.0f;
                    n_burn[n] = 0;
                } else {
                    int n_idx = nj * width + ni;
                    n_out_flags[n] = 0;
                    n_indices[n] = n_idx;
                    n_elev[n] = elevation[n_idx];
                    n_fwi[n] = fwi[n_idx];
                    n_asp[n] = aspect[n_idx];
                    n_veg[n] = vegetation_type[n_idx];
                    n_burn[n] = burnable[n_idx];
                    ++local_processed_cells;
                }

                uint8_t burnable_mask = (!burned_bin[n_indices[n]] && n_burn[n]);
                uint8_t valid_mask = !n_out_flags[n] && burnable_mask;
                n_upper[n] = valid_mask * upper_limit;
            }

            float n_probs[8];
            spread_probability_device(
                elev_c, wind_c,
                n_elev, n_veg, n_fwi, n_asp, n_upper,
                params, distance, elevation_mean, elevation_sd,
                n_probs
            );

            for (int n = 0; n < 8; ++n) {
                if (!burned_bin[n_indices[n]]) {
                    float rnd = hiprand_uniform(&local_state);
                    if (rnd < n_probs[n]) {
                        if (atomicExch(&burned_bin[n_indices[n]], 1) == 0) {
                            if (atomicCAS(&in_next_frontier[n_indices[n]], 0, 1) == 0) {
                                int pos = atomicAdd(next_frontier_count, 1);
                                next_frontier_0[pos] = n_coords_0[n];
                                next_frontier_1[pos] = n_coords_1[n];
                            }
                        }
                    }
                }
            }
            rng_states[center_idx] = local_state;
        }


        __syncthreads();

        for (int idx = tid; idx < width * height; idx += gridDim.x * blockDim.x) {
            in_next_frontier[idx] = 0;
        }

        __syncthreads();

        if (tid == 0) {
            int count = *next_frontier_count;
            *frontier_size = count;
            *next_frontier_count = 0;
            *done_flag = (count == 0);
        }

        __syncthreads();

        // Swap buffers (simple double buffer swap)
        int* tmp0 = frontier_0;
        int* tmp1 = frontier_1;
        frontier_0 = next_frontier_0;
        frontier_1 = next_frontier_1;
        next_frontier_0 = tmp0;
        next_frontier_1 = tmp1;
    }

    if (local_processed_cells)
        atomicAdd(args.processed_cells, local_processed_cells);
}

Fire simulate_fire(
    LandscapeSoA landscape,
    size_t n_row, size_t n_col,
    const std::vector<std::pair<size_t, size_t>>& ignition_cells,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    int n_replicate,
    float upper_limit = 1.0f
) {
    const size_t MAX_CELLS = n_row * n_col;

    std::vector<int> h_burned_ids_0(MAX_CELLS, -1);
    std::vector<int> h_burned_ids_1(MAX_CELLS, -1);
    std::vector<int> burned_bin(MAX_CELLS, 0);

    for (size_t i = 0; i < ignition_cells.size(); ++i) {
        h_burned_ids_0[i] = ignition_cells[i].first;
        h_burned_ids_1[i] = ignition_cells[i].second;
        burned_bin[utils::INDEX(ignition_cells[i].first, ignition_cells[i].second, n_col)] = 1;
    }

    std::vector<size_t> burned_ids_steps;
    burned_ids_steps.push_back(ignition_cells.size());

    // Copy 'angles' and 'moves' to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_angles), h_angles, sizeof(h_angles));
    hipMemcpyToSymbol(HIP_SYMBOL(d_moves), h_moves, sizeof(h_moves));

    int *d_frontier_0, *d_frontier_1;
    int *d_next_frontier_0, *d_next_frontier_1;
    int *d_frontier_size, *d_next_frontier_count, *d_done_flag;
    int *d_burned_bin;
    unsigned int *d_processed_cells;

    // Allocate memory for frontiers, flags and counters
    hipMalloc(&d_frontier_0, MAX_CELLS * sizeof(int));
    hipMalloc(&d_frontier_1, MAX_CELLS * sizeof(int));
    hipMalloc(&d_next_frontier_0, MAX_CELLS * sizeof(int));
    hipMalloc(&d_next_frontier_1, MAX_CELLS * sizeof(int));
    hipMalloc(&d_frontier_size, sizeof(int));
    hipMalloc(&d_next_frontier_count, sizeof(int));
    hipMalloc(&d_done_flag, sizeof(int));
    hipMalloc(&d_burned_bin, MAX_CELLS * sizeof(int));
    hipMalloc(&d_processed_cells, sizeof(unsigned int));

    int init_size = ignition_cells.size();
    hipMemcpy(d_frontier_0, h_burned_ids_0.data(), ignition_cells.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_1, h_burned_ids_1.data(), ignition_cells.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_size, &init_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_next_frontier_count, 0, sizeof(int));
    hipMemset(d_done_flag, 0, sizeof(int));
    hipMemcpy(d_burned_bin, burned_bin.data(), MAX_CELLS * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_processed_cells, 0, sizeof(unsigned int));

    // Copy landscape
    float *d_elevation, *d_fwi, *d_aspect, *d_wind_dir, *d_vegetation_type;
    uint8_t *d_burnable;
    hipMalloc(&d_elevation, MAX_CELLS * sizeof(float));
    hipMalloc(&d_fwi, MAX_CELLS * sizeof(float));
    hipMalloc(&d_aspect, MAX_CELLS * sizeof(float));
    hipMalloc(&d_wind_dir, MAX_CELLS * sizeof(float));
    hipMalloc(&d_vegetation_type, MAX_CELLS * sizeof(float));
    hipMalloc(&d_burnable, MAX_CELLS * sizeof(uint8_t));

    hipMemcpy(d_elevation, landscape.elevation.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fwi, landscape.fwi.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_aspect, landscape.aspect.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wind_dir, landscape.wind_dir.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vegetation_type, landscape.vegetation_type.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_burnable, landscape.burnable.data(), MAX_CELLS * sizeof(uint8_t), hipMemcpyHostToDevice);

    int *d_in_next_frontier;
    hipMalloc(&d_in_next_frontier, MAX_CELLS * sizeof(int));
    hipMemset(d_in_next_frontier, 0, MAX_CELLS * sizeof(int));

    // Copy simulation params
    SimulationParams* d_params;
    hipMalloc(&d_params, sizeof(SimulationParams));
    hipMemcpy(d_params, &params, sizeof(SimulationParams), hipMemcpyHostToDevice);

    // Prepare kernel arguments
    FireKernelParams args = {
        d_elevation, d_fwi, d_aspect, d_wind_dir, d_vegetation_type,
        d_burnable, d_burned_bin,
        static_cast<int>(n_col), static_cast<int>(n_row),
        d_processed_cells,
        d_params,
        distance, upper_limit, elevation_mean, elevation_sd,
        123 + n_replicate
    };

    // Launch the kernels
    int threads = 512;
    int blocks = (MAX_CELLS + threads - 1) / threads;

    // Initialize RNG states
    hiprandState* d_rng_states;
    hipMalloc(&d_rng_states, MAX_CELLS * sizeof(hiprandState));
    init_rng_kernel<<<blocks, threads>>>(d_rng_states, n_col, n_row, 123 + n_replicate);

    // Initialize the fire simulation
    double start_time = omp_get_wtime();
    fire_persistent_kernel<<<blocks, threads>>>(
        args,
        d_frontier_0, d_frontier_1, d_frontier_size,
        d_next_frontier_0, d_next_frontier_1, d_next_frontier_count,
        d_done_flag,
        d_in_next_frontier,
        d_rng_states
    );
    hipDeviceSynchronize();

    if (hipGetLastError() != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    double end_time = omp_get_wtime();
    double time_taken = end_time - start_time;

    // Copy results
    hipMemcpy(burned_bin.data(), d_burned_bin, MAX_CELLS * sizeof(int), hipMemcpyDeviceToHost);
    std::vector<size_t> ids_0, ids_1;
    for (size_t j = 0; j < n_row; ++j) {
        for (size_t i = 0; i < n_col; ++i) {
            if (burned_bin[utils::INDEX(i, j, n_col)]) {
                ids_0.push_back(i);
                ids_1.push_back(j);
            }
        }
    }

    // Copy processed cells count
    unsigned int processed_cells;
    hipMemcpy(&processed_cells, d_processed_cells, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_frontier_0); hipFree(d_frontier_1);
    hipFree(d_next_frontier_0); hipFree(d_next_frontier_1);
    hipFree(d_frontier_size); hipFree(d_next_frontier_count); hipFree(d_done_flag);
    hipFree(d_burned_bin); hipFree(d_processed_cells);

    hipFree(d_elevation); hipFree(d_fwi); hipFree(d_aspect);
    hipFree(d_wind_dir); hipFree(d_vegetation_type); hipFree(d_burnable);
    hipFree(d_params);

    return Fire{
        n_col, n_row, processed_cells, time_taken,
        burned_bin, ids_0, ids_1, burned_ids_steps
    };
}