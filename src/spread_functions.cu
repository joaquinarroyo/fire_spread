#include "hip/hip_runtime.h"
#include "spread_functions.cuh"

#define _USE_MATH_DEFINES
#include <cmath>
#include <random>
#include <vector>
#include <omp.h>
#include <iostream>
#include <array>
#include <random>

#include "fires.hpp"
#include "landscape.hpp"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

struct FireKernelParams {
    const float* elevation;
    const float* fwi;
    const float* aspect;
    const float* wind_dir;
    const float* vegetation_type;
    const uint8_t* burnable;

    int* burned_bin;
    int width;
    int height;

    const int* burned_ids_0;
    const int* burned_ids_1;
    int burned_size;

    int* new_burned_ids_0;
    int* new_burned_ids_1;
    int* new_burned_count;

    float distance;
    float upper_limit;
    float elevation_mean;
    float elevation_sd;

    const SimulationParams* params;
};

constexpr float PIf = 3.1415927f;
float h_angles[8] = {
    PIf * 3 / 4, PIf, PIf * 5 / 4, PIf / 2,
    PIf * 3 / 2, PIf / 4, 0, PIf * 7 / 4
};
int h_moves[8][2] = {
    { -1, -1 }, { -1, 0 }, { -1, 1 }, { 0, -1 },
    { 0, 1 }, { 1, -1 }, { 1, 0 }, { 1, 1 }
};
__constant__ float d_angles[8];
__constant__ int d_moves[8][2];

// RNG simple por thread (XORShift32)
__device__ uint32_t xorshift32(uint32_t& state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

__device__ void spread_probability_device(
    float burning_elevation,
    float burning_wind_direction,
    const float* elevations,
    const float* vegetation_types,
    const float* fwis,
    const float* aspects,
    const float* upper_limits,
    const SimulationParams* params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float* probs_out // salida de 8 elementos
) {
    for (int n = 0; n < 8; n++) {
        float slope_term = __sinf(atanf((elevations[n] - burning_elevation) / distance));
        float wind_term = __cosf(d_angles[n] - burning_wind_direction);
        float elev_term = (elevations[n] - elevation_mean) / elevation_sd;

        float linpred = params->independent_pred;

        if ((int)vegetation_types[n] == SUBALPINE) {
            linpred += params->subalpine_pred;
        } else if ((int)vegetation_types[n] == WET) {
            linpred += params->wet_pred;
        } else if ((int)vegetation_types[n] == DRY) {
            linpred += params->dry_pred;
        }

        linpred += params->fwi_pred * fwis[n];
        linpred += params->aspect_pred * aspects[n];
        linpred += wind_term * params->wind_pred + elev_term * params->elevation_pred + slope_term * params->slope_pred;

        probs_out[n] = upper_limits[n] / (1.0f + __expf(-linpred));
    }
}

__global__ void fire_step_kernel(FireKernelParams args) {
    // Desempaquetar argumentos para facilitar el uso
    const float* elevation = args.elevation;
    const float* fwi = args.fwi;
    const float* aspect = args.aspect;
    const float* wind_dir = args.wind_dir;
    const float* vegetation_type = args.vegetation_type;
    const uint8_t* burnable = args.burnable;

    int* burned_bin = args.burned_bin;
    int width = args.width;
    int height = args.height;

    const int* burned_ids_0 = args.burned_ids_0;
    const int* burned_ids_1 = args.burned_ids_1;
    int burned_size = args.burned_size;

    int* new_burned_ids_0 = args.new_burned_ids_0;
    int* new_burned_ids_1 = args.new_burned_ids_1;
    int* new_burned_count = args.new_burned_count;

    float distance = args.distance;
    float upper_limit = args.upper_limit;
    float elevation_mean = args.elevation_mean;
    float elevation_sd = args.elevation_sd;

    const SimulationParams* params = args.params;

    // ---- Código original del kernel a partir de acá ----
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= burned_size) return;

    // Setup RNG con una semilla diferente por thread
    hiprandState state;
    hiprand_init(1234ULL, /* seed */
             idx,     /* sequence: hace que cada hilo tenga su propia secuencia */
             0,       /* offset */
             &state);

    int i = burned_ids_0[idx];
    int j = burned_ids_1[idx];

    int center_idx = j * width + i;

    float elev_c = elevation[center_idx];
    float wind_c = wind_dir[center_idx];

    float elevations[8], fwis[8], aspects[8], vegtypes[8], upper_limits[8];
    for (int n = 0; n < 8; ++n) {
        int ni = i + d_moves[n][0];
        int nj = j + d_moves[n][1];
        if (ni < 0 || nj < 0 || ni >= width || nj >= height) {
            upper_limits[n] = 0;
            continue;
        }

        int n_idx = nj * width + ni;

        elevations[n] = elevation[n_idx];
        fwis[n] = fwi[n_idx];
        aspects[n] = aspect[n_idx];
        vegtypes[n] = vegetation_type[n_idx];

        upper_limits[n] = (!burned_bin[n_idx] && burnable[n_idx]) ? upper_limit : 0.0f;
    }

    float probs[8];
    spread_probability_device(
        elev_c, wind_c, elevations, vegtypes, fwis, aspects, upper_limits,
        params, distance, elevation_mean, elevation_sd, probs
    );

    for (int n = 0; n < 8; ++n) {
        int ni = i + d_moves[n][0];
        int nj = j + d_moves[n][1];
        if (ni < 0 || nj < 0 || ni >= width || nj >= height) continue;
        int n_idx = nj * width + ni;

        float rand_val = hiprand_uniform(&state);
        if (rand_val < probs[n]) {
            if (atomicExch(&burned_bin[n_idx], 1) == 0) {
                int pos = atomicAdd(new_burned_count, 1);
                if (pos < width * height) {
                    new_burned_ids_0[pos] = ni;
                    new_burned_ids_1[pos] = nj;
                }
            }
        }
    }
}

Fire simulate_fire(
    LandscapeSoA landscape,
    size_t n_row, size_t n_col,
    const std::vector<std::pair<size_t, size_t>>& ignition_cells,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    int n_replicate,
    float upper_limit = 1.0f
) {
    const size_t MAX_BURNED_CELLS = n_row * n_col;
    n_replicate += 1;

    std::vector<int> h_burned_ids_0;
    std::vector<int> h_burned_ids_1;
    h_burned_ids_0.resize(MAX_BURNED_CELLS, -1);
    h_burned_ids_1.resize(MAX_BURNED_CELLS, -1);
    std::vector<int> burned_bin(n_row * n_col, 0);

    size_t end = ignition_cells.size();
    size_t burning_size = end;

    for (size_t i = 0; i < end; i++) {
        h_burned_ids_0[i] = ignition_cells[i].first;
        h_burned_ids_1[i] = ignition_cells[i].second;
        burned_bin[utils::INDEX(ignition_cells[i].first, ignition_cells[i].second, n_col)] = 1;
    }

    std::vector<size_t> burned_ids_steps;
    burned_ids_steps.push_back(end);

    hipMemcpyToSymbol(HIP_SYMBOL(d_angles), h_angles, sizeof(h_angles));
    hipMemcpyToSymbol(HIP_SYMBOL(d_moves), h_moves, sizeof(h_moves));
    
    // Punteros GPU
    int *d_burned_ids_0, *d_burned_ids_1;
    int *d_new_burned_ids_0, *d_new_burned_ids_1;
    int *d_burned_bin, *d_new_burned_count;

    // Reservar memoria en GPU
    hipMalloc(&d_burned_ids_0, MAX_BURNED_CELLS * sizeof(int));
    hipMalloc(&d_burned_ids_1, MAX_BURNED_CELLS * sizeof(int));
    hipMalloc(&d_new_burned_ids_0, MAX_BURNED_CELLS * sizeof(int));
    hipMalloc(&d_new_burned_ids_1, MAX_BURNED_CELLS * sizeof(int));
    hipMalloc(&d_burned_bin, n_row * n_col * sizeof(int));
    hipMalloc(&d_new_burned_count, sizeof(int));

    // Copiar datos iniciales a GPU
    hipMemcpy(d_burned_ids_0, h_burned_ids_0.data(), end * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_burned_ids_1, h_burned_ids_1.data(), end * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_burned_bin, burned_bin.data(), n_row * n_col * sizeof(int), hipMemcpyHostToDevice);

    // Reservar memoria en device
    float *d_elevation, *d_fwi, *d_aspect, *d_wind_dir, *d_vegetation_type;
    uint8_t *d_burnable;

    hipMalloc(&d_elevation, n_row * n_col * sizeof(float));
    hipMalloc(&d_fwi, n_row * n_col * sizeof(float));
    hipMalloc(&d_aspect, n_row * n_col * sizeof(float));
    hipMalloc(&d_wind_dir, n_row * n_col * sizeof(float));
    hipMalloc(&d_vegetation_type, n_row * n_col * sizeof(float));
    hipMalloc(&d_burnable, n_row * n_col * sizeof(uint8_t));

    // Copiar datos del host al device
    hipMemcpy(d_elevation, landscape.elevation.data(), n_row * n_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fwi, landscape.fwi.data(), n_row * n_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_aspect, landscape.aspect.data(), n_row * n_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wind_dir, landscape.wind_dir.data(), n_row * n_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vegetation_type, landscape.vegetation_type.data(), n_row * n_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_burnable, landscape.burnable.data(), n_row * n_col * sizeof(uint8_t), hipMemcpyHostToDevice);

    SimulationParams* d_params;
    hipMalloc(&d_params, sizeof(SimulationParams));
    hipMemcpy(d_params, &params, sizeof(SimulationParams), hipMemcpyHostToDevice);

    // Comenzar simulación
    double start_time = omp_get_wtime();
    unsigned int processed_cells = 0;

    hipMemcpy(d_burned_ids_0, h_burned_ids_0.data(), end * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_burned_ids_1, h_burned_ids_1.data(), end * sizeof(int), hipMemcpyHostToDevice);

    int* current_burned_ids_0 = d_burned_ids_0;
    int* current_burned_ids_1 = d_burned_ids_1;
    int* next_burned_ids_0 = d_new_burned_ids_0;
    int* next_burned_ids_1 = d_new_burned_ids_1;

    while (burning_size > 0) {
        int threads = 256;
        int blocks = (burning_size + threads - 1) / threads;

        hipMemset(d_new_burned_count, 0, sizeof(int));

        FireKernelParams args = {
            d_elevation, d_fwi, d_aspect, d_wind_dir, d_vegetation_type, d_burnable,
            d_burned_bin, static_cast<int>(n_col), static_cast<int>(n_row),
            current_burned_ids_0, current_burned_ids_1,
            static_cast<int>(burning_size),
            next_burned_ids_0, next_burned_ids_1, d_new_burned_count,
            distance, upper_limit, elevation_mean, elevation_sd,
            d_params
        };

        fire_step_kernel<<<blocks, threads>>>(args);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("[CUDA ERROR] %s\n", hipGetErrorString(err));
            break;
        }

        int new_count;
        hipMemcpy(&new_count, d_new_burned_count, sizeof(int), hipMemcpyDeviceToHost);

        // Copiar los nuevos IDs al host al final (sin memcpy hacia device)
        hipMemcpy(h_burned_ids_0.data() + end, next_burned_ids_0, new_count * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_burned_ids_1.data() + end, next_burned_ids_1, new_count * sizeof(int), hipMemcpyDeviceToHost);

        end += new_count;
        burning_size = new_count;
        processed_cells += new_count;
        burned_ids_steps.push_back(end);

        // Swap de buffers
        std::swap(current_burned_ids_0, next_burned_ids_0);
        std::swap(current_burned_ids_1, next_burned_ids_1);
    }

    double end_time = omp_get_wtime();
    double time_taken = end_time - start_time;

    hipMemcpy(burned_bin.data(), d_burned_bin, n_row * n_col * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria
    hipFree(d_burned_ids_0);
    hipFree(d_burned_ids_1);
    hipFree(d_new_burned_ids_0);
    hipFree(d_new_burned_ids_1);
    hipFree(d_burned_bin);
    hipFree(d_new_burned_count);

    hipFree(d_elevation);
    hipFree(d_fwi);
    hipFree(d_aspect);
    hipFree(d_wind_dir);
    hipFree(d_vegetation_type);
    hipFree(d_burnable);

    std::vector<size_t> ids_0_size_t(h_burned_ids_0.begin(), h_burned_ids_0.begin() + end);
    std::vector<size_t> ids_1_size_t(h_burned_ids_1.begin(), h_burned_ids_1.begin() + end);

    return Fire{
      n_col, n_row, processed_cells, time_taken,
      burned_bin, ids_0_size_t, ids_1_size_t, burned_ids_steps
    };
}
